
#include <hip/hip_runtime.h>
#include <iostream>

#define N 512

// CUDA 内核函数：每个线程处理一个数组元素
__global__ void vector_add(float* A, float* B, float* C) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    // CPU 上的数组
    float h_A[N], h_B[N], h_C[N];

    // 初始化数据
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // GPU 上的数组指针
    float *d_A, *d_B, *d_C;

    // 分配 GPU 内存
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    // 复制数据到 GPU
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // 启动 kernel，每个 block 256 个线程
    vector_add<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C);

    // 把结果从 GPU 拷贝回来
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // 打印部分结果
    for (int i = 0; i < 10; ++i) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // 释放 GPU 内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}