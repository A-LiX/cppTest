#include <iostream>
#include <hip/hip_runtime.h>

#define N 512

// CUDA 内核函数：每个线程处理一个数组元素
__global__ void vector_add(float* A, float* B, float* C) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

// CUDA 错误检查函数
void checkCudaError(hipError_t error) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}

int main() {
    // CPU 上的数组
    float h_A[N], h_B[N], h_C[N];

    // 初始化数据
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // GPU 上的数组指针
    float *d_A, *d_B, *d_C;

    // 分配 GPU 内存
    checkCudaError(hipMalloc((void**)&d_A, N * sizeof(float)));
    checkCudaError(hipMalloc((void**)&d_B, N * sizeof(float)));
    checkCudaError(hipMalloc((void**)&d_C, N * sizeof(float)));

    // 复制数据到 GPU
    checkCudaError(hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice));

    // 启动 kernel，每个 block 256 个线程
    vector_add<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C);

    // 检查 kernel 启动后的错误
    checkCudaError(hipGetLastError());
    
    // 等待 GPU 完成计算
    checkCudaError(hipDeviceSynchronize());

    // 把结果从 GPU 拷贝回来
    checkCudaError(hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    // 打印部分结果
    for (int i = 0; i < 10; ++i) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << std::endl;
    }

    // 释放 GPU 内存
    checkCudaError(hipFree(d_A));
    checkCudaError(hipFree(d_B));
    checkCudaError(hipFree(d_C));

    return 0;
}